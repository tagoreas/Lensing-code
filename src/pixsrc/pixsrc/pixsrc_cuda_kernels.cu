#include "hip/hip_runtime.h"
// -*- C++ -*-
// the previous line is for emacs .. so that it formats
// this file according to the c++ format style
// specified in ~/.emacs

// this is CUDA code that needs to be compiled by NVCC
// the resulting ibject file can be linked with g++
// I think most of this code is unused currently.

#ifndef PS_FPT
#ifdef SINGLE_PRECISION
#define PS_FPT float
#endif
#ifdef DOUBLE_PRECISION
#define PS_FPT double
#endif
#endif

#include <hipsparse.h>
#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include <cusp/csr_matrix.h>
#include <cusp/krylov/cg.h>
#include <cusp/precond/ainv.h>
#include <cusp/precond/aggregation/smoothed_aggregation.h>
#include <cusp/krylov/gmres.h>
#include <cusp/print.h>


// fill vector with constant value
__global__ void psc_fill_kernel( PS_FPT *vec, PS_FPT val )
{
    int i = blockIdx.x;

    vec[i] = val;

    return;
}

__global__ void psc_cpy_mat_col_to_vec_kernel( int col,
                                               int *apc, int *aic, PS_FPT *axc, PS_FPT *vec )
{
    for( int g=apc[col]; g<apc[col+1]; ++g )
        vec[aic[g]] = axc[g];

    return;
}

__global__ void psc_undo_cpy_mat_col_to_vec_kernel( int col,
                                                    int *apc, int *aic, PS_FPT *vec )
{
    for( int g=apc[col]; g<apc[col+1]; ++g )
        vec[aic[g]] = 0;

    return;
}

__global__ void psc_cpy_mat_col_to_vec_kernel( int col, int nrow,
                                               int *apc, int *aic, PS_FPT *axc, PS_FPT *vec_ )
{
    int i = blockIdx.x;

    col = i;
    PS_FPT *vec = vec_ + i*nrow;

    for( int g=apc[col]; g<apc[col+1]; ++g )
        vec[aic[g]] = axc[g];

    return;
}

__global__ void psc_cpy_mat_diag_to_vec_kernel( int col, int nrow,
                                                int *apc, int *aic, PS_FPT *axc, PS_FPT *vec_ )
{
    int i = blockIdx.x;

    col = i;
    PS_FPT *vec = vec_;

    for( int g=apc[col]; g<apc[col+1]; ++g )
        if( col == aic[g])
            vec[aic[g]] = axc[g];

    return;
}

__global__ void sum_vec_for_trace( PS_FPT *vec, int l_side )
{
    for( int g=1; g<l_side; ++g )
        vec[0] += vec[l_side*g+g];

    return;
}









template <typename Monitor>
void report_status(Monitor& monitor)
{
    if (monitor.converged())
    {
        std::cout << "Solver converged to " << monitor.tolerance() << " tolerance";
        std::cout << " after " << monitor.iteration_count() << " iterations";
        std::cout << " (" << monitor.residual_norm() << " final residual)" << std::endl;
    }
    else
    {
        std::cout << "Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
        std::cout << " to " << monitor.tolerance() << " tolerance ";
        std::cout << " (" << monitor.residual_norm() << " final residual)" << std::endl;
    }
}









void psc_fill( PS_FPT *vec, PS_FPT val, int size, hipStream_t *str )
{
    psc_fill_kernel<<<size,1,0,*str>>>( vec, val );
/*
  thrust::device_ptr<PS_FPT> dev_ptr( vec );
  thrust::fill( dev_ptr, dev_ptr + (size_t)size, val );
*/
}

void psc_cpy_mat_col_to_vec( int col, int *apc, int *aic, PS_FPT *axc, PS_FPT *vec )
{
    psc_cpy_mat_col_to_vec_kernel <<< 1,1 >>> ( col, apc, aic, axc, vec );

    //for( int g=apc[col]; g<apc[col+1]; ++g )
    //     vec[aic[g]] = axc[g];
}

void psc_undo_cpy_mat_col_to_vec( int col, int *apc, int *aic, PS_FPT *vec )
{
    psc_undo_cpy_mat_col_to_vec_kernel <<< 1,1 >>> ( col, apc, aic, vec );

    //for( int g=apc[col]; g<apc[col+1]; ++g )
    //     vec[aic[g]] = 0;
}

PS_FPT psc_sum_vector( PS_FPT *vec, int size )
{
    thrust::device_ptr<PS_FPT> dev_ptr( vec );
    return thrust::reduce( dev_ptr, dev_ptr + (size_t)size,
                           (PS_FPT)0, thrust::plus<PS_FPT>() );
}

void psc_tri_eq_solve_cusp( int *p, int *i, PS_FPT *x,
                            PS_FPT *res_, PS_FPT *rhs_, int l_side, int nnz, void *prec )
{
    prec = 0;

    // typedef's
    typedef typename cusp::array1d_view< thrust::device_ptr<int>    > DeviceIndexArrayView;
    typedef typename cusp::array1d_view< thrust::device_ptr<PS_FPT> > DeviceValueArrayView;

    typedef cusp::csr_matrix_view<DeviceIndexArrayView,
        DeviceIndexArrayView,
        DeviceValueArrayView> DeviceView;

    // wrapping matrix

    thrust::device_ptr<int>    wrapped_device_Ap( p );
    thrust::device_ptr<int>    wrapped_device_Aj( i );
    thrust::device_ptr<PS_FPT> wrapped_device_Ax( x );

    DeviceIndexArrayView row_offsets   ( wrapped_device_Ap, wrapped_device_Ap + l_side+1 );
    DeviceIndexArrayView column_indices( wrapped_device_Aj, wrapped_device_Aj + nnz      );
    DeviceValueArrayView values        ( wrapped_device_Ax, wrapped_device_Ax + nnz      );

    DeviceView mat( l_side, l_side, nnz, row_offsets, column_indices, values );

    //cusp::print(mat);

    // wrapping vectors

    thrust::device_ptr<PS_FPT> wrapped_device_rhs( rhs_ );
    thrust::device_ptr<PS_FPT> wrapped_device_res( res_ );

    DeviceValueArrayView rhs( wrapped_device_rhs, wrapped_device_rhs + l_side );
    DeviceValueArrayView res( wrapped_device_res, wrapped_device_res + l_side );

    // set stopping criteria
    //  iteration_limit    = 1000
    //  relative_tolerance = 1e-3
    cusp::default_monitor<PS_FPT> monitor(rhs, 1000, 1e-6);

    // cusp::precond::scaled_bridson_ainv<PS_FPT, cusp::device_memory> M(mat, .1);
    // cusp::krylov::cg(mat, res, rhs, monitor ,M );
    // cusp::krylov::cg(mat, res, rhs, monitor /*,M*/ );

    if( prec )
    {
        cusp::precond::scaled_bridson_ainv<PS_FPT, cusp::device_memory> *preccer =
            (cusp::precond::scaled_bridson_ainv<PS_FPT, cusp::device_memory>*) prec;
        cusp::krylov::cg(mat, res, rhs, monitor , *preccer );
    }
    else
    {
        cusp::krylov::cg(mat, res, rhs, monitor /*,M*/ );
    }

    //report_status(monitor);
}

void psc_destr_preconditioner_cusp( void *prec )
{
    typedef typename cusp::precond::scaled_bridson_ainv<PS_FPT, cusp::device_memory> cusp_prec;
    cusp_prec *preccer = (cusp_prec*)prec;

    delete preccer;
    prec = 0;
}

void psc_get_preconditioner_cusp( int *p, int *i, PS_FPT *x,
                                  int l_side, int nnz, void **prec )
{


    // typedef's

    typedef typename cusp::array1d_view< thrust::device_ptr<int>    > DeviceIndexArrayView;
    typedef typename cusp::array1d_view< thrust::device_ptr<PS_FPT> > DeviceValueArrayView;

    typedef cusp::csr_matrix_view<DeviceIndexArrayView,
        DeviceIndexArrayView,
        DeviceValueArrayView> DeviceView;

    typedef typename cusp::precond::scaled_bridson_ainv<PS_FPT, cusp::device_memory> cusp_prec;

    // wrapping matrix

    thrust::device_ptr<int>    wrapped_device_Ap( p );
    thrust::device_ptr<int>    wrapped_device_Aj( i );
    thrust::device_ptr<PS_FPT> wrapped_device_Ax( x );

    DeviceIndexArrayView row_offsets   ( wrapped_device_Ap, wrapped_device_Ap + l_side+1 );
    DeviceIndexArrayView column_indices( wrapped_device_Aj, wrapped_device_Aj + nnz      );
    DeviceValueArrayView values        ( wrapped_device_Ax, wrapped_device_Ax + nnz      );

    DeviceView mat( l_side, l_side, nnz, row_offsets, column_indices, values );

    cusp_prec *M = new cusp_prec(mat, 0.1);

    *prec = (void*)M;
}








float tot=0;
class matrixfree_op : public cusp::linear_operator<PS_FPT,cusp::device_memory>
{
public:
    typedef cusp::linear_operator<PS_FPT,cusp::device_memory> super;

    int N;
    hipsparseHandle_t   *handle;
    hipsparseMatDescr_t *descr_a;
    int *ap, *ai;
    int a_nnz;
    PS_FPT *ax;

    // constructor
    matrixfree_op( int N, void *handle_, void *descr_, int *ap_, int *ai_, PS_FPT *ax_, int a_nnz_ )
        : super(N*N,N*N),
          N(N),
          handle ((hipsparseHandle_t*  )handle_),
          descr_a((hipsparseMatDescr_t*)descr_ ),
          ap(ap_), ai(ai_), ax(ax_), a_nnz(a_nnz_) {}

    template <typename VectorType1,
              typename VectorType2>
    void operator()(const VectorType1& x, VectorType2& y) const
        {
            // obtain a raw pointer to device memory
            const PS_FPT* x_ptr = thrust::raw_pointer_cast(&x[0]);
            PS_FPT* y_ptr       = thrust::raw_pointer_cast(&y[0]);

            hipsparseOperation_t trans_a = HIPSPARSE_OPERATION_NON_TRANSPOSE;
            PS_FPT dummy_one  = 1;
            PS_FPT dummy_zero = 0;

            hipEvent_t evt[2];
            hipEventCreate( &evt[0] );
            hipEventCreate( &evt[1] );
            hipEventRecord( evt[0], 0 );

            for( int j=0; j<N; ++j )
            {
                //hipsparseSetStream( *handle, stream[str] );

#ifdef SINGLE_PRECISION
                hipsparseScsrmv( *handle, trans_a,
                                N, N, a_nnz, &dummy_one,
                                *descr_a,
                                ax, ap, ai,
                                x_ptr+j*N,
                                &dummy_zero, y_ptr+j*N        );
#else
                hipsparseDcsrmv( *handle, trans_a,
                                N, N, a_nnz, &dummy_one,
                                *descr_a,
                                ax, ap, ai,
                                x_ptr+j*N,
                                &dummy_zero, y_ptr+j*N        );
#endif

            }

            hipEventRecord( evt[1], 0 );
            hipEventSynchronize( evt[1] );
            float elapsedTime;
            hipEventElapsedTime( &elapsedTime, evt[0], evt[1] );
            hipEventDestroy( evt[0] );
            hipEventDestroy( evt[1] );
            tot += elapsedTime;
//          std::cout << "took " << elapsedTime << " ms" << std::endl;

        }
};

void psc_tr_inv_a_b_matrixfree( int *ap, int *ai, PS_FPT *ax,
                                int *cp, int *ci, PS_FPT *cx,
                                PS_FPT *res, PS_FPT *rhs, int l_side,
                                void *handle, void *descr, int a_nnz, void *prec  )
{
    hipEvent_t evt[2];
    hipEventCreate( &evt[0] );
    hipEventCreate( &evt[1] );
    hipEventRecord( evt[0], 0 );

    int l_side2 = l_side * l_side;

    // typedef's
    typedef typename cusp::array1d_view< thrust::device_ptr<PS_FPT> > DeviceValueArrayView;

    // zero out arrays
    thrust::device_ptr<PS_FPT> res_wrapped( res );
    thrust::device_ptr<PS_FPT> rhs_wrapped( rhs );
    thrust::fill( res_wrapped, res_wrapped + l_side, 1 );
    thrust::fill( rhs_wrapped, rhs_wrapped + l_side, 0 );

    // fill in rhs vector
    psc_cpy_mat_col_to_vec_kernel<<<l_side,1>>>( -1, l_side, cp, ci, cx, rhs );

    // wrapping vectors
    DeviceValueArrayView res_wrapped2( res_wrapped, res_wrapped + l_side2 );
    DeviceValueArrayView rhs_wrapped2( rhs_wrapped, rhs_wrapped + l_side2 );

    // linear operator
    matrixfree_op op( l_side, handle, descr, ap, ai, ax, a_nnz );

    cusp::default_monitor<PS_FPT> monitor(rhs_wrapped2, 1000, 1e-3);

    if( prec )
    {
        cusp::precond::scaled_bridson_ainv<PS_FPT, cusp::device_memory> *preccer =
            (cusp::precond::scaled_bridson_ainv<PS_FPT, cusp::device_memory>*) prec;
        cusp::krylov::cg(op, res_wrapped2, rhs_wrapped2, monitor , *preccer );
    }
    else
    {
        cusp::krylov::cg(op, res_wrapped2, rhs_wrapped2, monitor);
    }

    sum_vec_for_trace<<<1,1>>>( res, l_side );

    hipEventRecord( evt[1], 0 );
    hipEventSynchronize( evt[1] );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, evt[0], evt[1] );
    hipEventDestroy( evt[0] );
    hipEventDestroy( evt[1] );
    std::cout << "total took " << elapsedTime << " ms " << tot << std::endl;
    //exit(1);
}





/*
  class matrixfree_op : public cusp::linear_operator<PS_FPT,cusp::device_memory>
  {
  public:
  typedef cusp::linear_operator<PS_FPT,cusp::device_memory> super;

  int N;
  hipsparseHandle_t   *handle;
  hipsparseMatDescr_t *descr_a;
  int *ap, *ai;
  int a_nnz;
  PS_FPT *ax;

  // constructor
  matrixfree_op( int N, void *handle_, void *descr_, int *ap_, int *ai_, PS_FPT *ax_, int a_nnz_ )
  : super(N*N,N*N),
  N(N),
  handle ((hipsparseHandle_t*  )handle_),
  descr_a((hipsparseMatDescr_t*)descr_ ),
  ap(ap_), ai(ai_), ax(ax_), a_nnz(a_nnz_) {}

  template <typename VectorType1,
  typename VectorType2>
  void operator()(const VectorType1& x, VectorType2& y) const
  {
  // obtain a raw pointer to device memory
  const PS_FPT* x_ptr = thrust::raw_pointer_cast(&x[0]);
  PS_FPT* y_ptr       = thrust::raw_pointer_cast(&y[0]);

  hipsparseOperation_t trans_a = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  PS_FPT dummy_one  = 1;
  PS_FPT dummy_zero = 0;

  for( int j=0; j<N; ++j )
  {
  //hipsparseSetStream( *handle, stream[str] );

  #ifdef SINGLE_PRECISION
  hipsparseScsrmv( *handle, trans_a,
  N, N, a_nnz, &dummy_one,
  *descr_a,
  ax, ap, ai,
  x_ptr+j*N,
  &dummy_zero, y_ptr+j*N        );
  #else
  hipsparseDcsrmv( *handle, trans_a,
  N, N, a_nnz, &dummy_one,
  *descr_a,
  ax, ap, ai,
  x_ptr+j*N,
  &dummy_zero, y_ptr+j*N        );
  #endif

  }

  }
  };

  void psc_tr_inv_a_b_matrixfree( int *ap, int *ai, PS_FPT *ax,
  int *cp, int *ci, PS_FPT *cx,
  PS_FPT *res, PS_FPT *rhs, int l_side,
  void *handle, void *descr, int a_nnz  )
  {
  int l_side2 = l_side * l_side;

  // typedef's
  typedef typename cusp::array1d_view< thrust::device_ptr<PS_FPT> > DeviceValueArrayView;

  // zero out arrays
  psc_fill( res, 0, l_side2 );
  psc_fill( rhs, 0, l_side2 );

  // fill in rhs vector
  psc_cpy_mat_col_to_vec_kernel<<<l_side,1>>>( -1, l_side, cp, ci, cx, rhs );

  // wrapping vectors
  thrust::device_ptr<PS_FPT> res_wrapped( res );
  thrust::device_ptr<PS_FPT> rhs_wrapped( rhs );
  DeviceValueArrayView res_wrapped2( res_wrapped, res_wrapped + l_side2 );
  DeviceValueArrayView rhs_wrapped2( rhs_wrapped, rhs_wrapped + l_side2 );

  // linear operator
  matrixfree_op op( l_side, handle, descr, ap, ai, ax, a_nnz );

  cusp::verbose_monitor<PS_FPT> monitor(rhs_wrapped2, 1000, 1e-3);

  cusp::krylov::cg(op, res_wrapped2, rhs_wrapped2, monitor);

  sum_vec_for_trace<<<1,1>>>( res, l_side );
  }

*/
